// ----------------------------------------------------------------------
//
// Copyright (c) 2011, 2014, Gerhard Zumbusch.
// All rights reserved.
//
// ----------------------------------------------------------------------

// find CUDA compute capabilities
// compile with nvcc


#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
  int dev = 0;
  if (hipSetDevice(dev)) return 1;
  hipDeviceProp_t prop;
  if (hipGetDeviceProperties(&prop, dev)) return 1;
  printf("-arch=sm_%d%d", prop.major, prop.minor);
  hipDeviceReset();
  return 0;
}
